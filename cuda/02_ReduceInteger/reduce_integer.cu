#include "hip/hip_runtime.h"
#include "freshman.h"
#include <hip/hip_runtime.h>

int recursiveReduce(int *arr, size_t const size) {
    if (size == 1) { return arr[0]; }
    size_t stride = size / 2;
    for (size_t i = 0; i < stride; ++i) { arr[i] += arr[i + stride]; }
    if (size % 2 == 1) { arr[0] += arr[size - 1]; }
    return recursiveReduce(arr, stride);
}

__global__ void warmup(int *iarr, int *oarr, size_t n) {
    // Get the threadid, we are computing in current block
    unsigned int tid = threadIdx.x;
    unsigned int idx = tid + blockDim.x * blockIdx.x;
    if (idx >= n) return;

    // Get current block's first index
    int *larr = iarr + blockIdx.x * blockDim.x;

    // Compute the sum in each block seperately
    // Reduce sum with increasing stride
    for (int stride = 1; stride < blockDim.x; stride *= 2) {
        if ((tid % (2 * stride)) == 0)
            larr[tid] += tid + stride >= blockDim.x ? 0 : larr[tid + stride];
        __syncthreads();
    }
    if (tid == 0) oarr[blockIdx.x] = larr[0];
}

__global__ void reduceNeighbored(int *iarr, int *oarr) {
    unsigned int tid = threadIdx.x;

    int *larr = iarr + blockIdx.x * blockDim.x;

    for (int stride = 1; stride < blockDim.x; stride *= 2) {
        if ((tid % (2 * stride)) == 0)
            larr[tid] += tid + stride >= blockDim.x ? 0 : larr[tid + stride];
        __syncthreads();
    }
    if (tid == 0) oarr[blockIdx.x] = larr[0];
}

__global__ void reduceNeighboredLess(int *iarr, int *oarr) {
    unsigned int tid = threadIdx.x;

    int *larr = iarr + blockIdx.x * blockDim.x;
    for (int stride = 1; stride < blockDim.x; stride *= 2) {
        int index = 2 * stride * tid;
        if (index < blockDim.x) {
            larr[index] +=
                    index + stride >= blockDim.x ? 0 : larr[index + stride];
        }
        __syncthreads();
    }
    if (tid == 0) oarr[blockIdx.x] = larr[0];
}

__global__ void reduceInterLeaved(int *iarr, int *oarr, unsigned int n) {
    unsigned int tid = threadIdx.x;
    unsigned int idx = blockIdx.x * blockDim.x + threadIdx.x;

    int *larr = iarr + blockIdx.x * blockDim.x;
    if (idx >= n) return;

    for (int stride = blockDim.x / 2; stride > 0; stride >>= 1) {
        if (tid < stride) larr[tid] += larr[tid + stride];
        __syncthreads();
    }

    if (tid == 0) oarr[blockIdx.x] = larr[0];
}

int main(int argc, char **argv) {
    initDevice(0);
    int size = 1 << 24;
    printf(" with array size %d \n", size);

    // Compute execution configuration
    int blockSize = 1024;
    if (argc > 1) { blockSize = atoi(argv[1]); }
    dim3 block(blockSize, 1);
    dim3 grid((size - 1) / block.x + 1, 1);
    printf(" grid %d block %d \n", grid.x, block.x);

    // Allocate host memory
    size_t bytes = size * sizeof(int);
    int *iarr_host = (int *) malloc(bytes);
    int *oarr_host = (int *) malloc(grid.x * sizeof(int));
    int *tmp = (int *) malloc(bytes);

    // Initialize the array
    initializeArr(iarr_host, size);

    memcpy(tmp, iarr_host, bytes);
    double iStart, iElaps;

    // Allocate device memory
    int *iarr_dev = NULL;
    int *oarr_dev = NULL;
    CHECK(hipMalloc((void **) &iarr_dev, bytes));
    CHECK(hipMalloc((void **) &oarr_dev, grid.x * sizeof(int)));

    // CPU reduction
    int cpu_sum = 0;
    iStart = cpuSecond();
    // for (int i = 0; i < size; ++i) { cpu_sum += tmp[i]; }
    // This recursive algorithm is much faster then for loop sum above
    // since it reduces sum times
    cpu_sum = recursiveReduce(tmp, size);
    iElaps = cpuSecond() - iStart;
    printf(" cpu reduce elapsed %lf ms cpu_sum: %d \n", iElaps, cpu_sum);

    int gpu_sum = 0;
    // Kernel 0: Warm up
    CHECK(hipMemcpy(iarr_dev, iarr_host, bytes, hipMemcpyHostToDevice));
    CHECK(hipDeviceSynchronize());
    iStart = cpuSecond();
    warmup<<<grid, block>>>(iarr_dev, oarr_dev, size);
    CHECK(hipDeviceSynchronize());
    CHECK(hipMemcpy(oarr_host, oarr_dev, grid.x * sizeof(int),
                     hipMemcpyDeviceToHost));
    gpu_sum = 0;
    for (int i = 0; i < grid.x; ++i) { gpu_sum += oarr_host[i]; }
    iElaps = cpuSecond() - iStart;
    printf(" gpu warmup elapsed %lf ms gpu_sum: %d <<<grid %d block "
           "%d>>>\n",
           iElaps, gpu_sum, grid.x, block.x);

    // Kernel 1: Reduce neighbored (same as warm up above)
    // TODO: actually the max warp size is 32, and our code is not working for not 2^n block size which is greater than 32
    CHECK(hipMemcpy(iarr_dev, iarr_host, bytes, hipMemcpyHostToDevice));
    CHECK(hipDeviceSynchronize());
    iStart = cpuSecond();
    warmup<<<grid, block>>>(iarr_dev, oarr_dev, size);
    CHECK(hipDeviceSynchronize());
    CHECK(hipMemcpy(oarr_host, oarr_dev, grid.x * sizeof(int),
                     hipMemcpyDeviceToHost));
    gpu_sum = 0;
    for (int i = 0; i < grid.x; ++i) { gpu_sum += oarr_host[i]; }
    iElaps = cpuSecond() - iStart;
    printf(" gpu neighbored elapsed %lf ms gpu_sum: %d <<<grid %d block "
           "%d>>>\n",
           iElaps, gpu_sum, grid.x, block.x);

    // Kernel 2: Reduce neighbored less
    CHECK(hipMemcpy(iarr_dev, iarr_host, bytes, hipMemcpyHostToDevice));
    CHECK(hipDeviceSynchronize());
    iStart = cpuSecond();
    reduceNeighboredLess<<<grid, block>>>(iarr_dev, oarr_dev);
    CHECK(hipDeviceSynchronize());
    CHECK(hipMemcpy(oarr_host, oarr_dev, grid.x * sizeof(int),
                     hipMemcpyDeviceToHost));
    gpu_sum = 0;
    for (int i = 0; i < grid.x; ++i) gpu_sum += oarr_host[i];
    iElaps = cpuSecond() - iStart;
    printf(" gpu neighbored less elapsed %lf ms gpu_sum: %d <<<grid %d block "
           "%d>>>\n",
           iElaps, gpu_sum, grid.x, block.x);

    // Kernel 3: Reduce neighbored less
    CHECK(hipMemcpy(iarr_dev, iarr_host, bytes, hipMemcpyHostToDevice));
    CHECK(hipDeviceSynchronize());
    iStart = cpuSecond();
    reduceInterLeaved<<<grid, block>>>(iarr_dev, oarr_dev, size);
    CHECK(hipDeviceSynchronize());
    CHECK(hipMemcpy(oarr_host, oarr_dev, grid.x * sizeof(int),
                     hipMemcpyDeviceToHost));
    gpu_sum = 0;
    for (int i = 0; i < grid.x; ++i) gpu_sum += oarr_host[i];
    iElaps = cpuSecond() - iStart;
    printf(" gpu reduce inter leaved less elapsed %lf ms gpu_sum: %d <<<grid "
           "%d block "
           "%d>>>\n",
           iElaps, gpu_sum, grid.x, block.x);
}